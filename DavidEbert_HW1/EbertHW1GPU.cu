/*
David Ebert
Homework 1 - GPU Addition

Output:
(N=100)
Time in milliseconds= 0.053000000000000
Last Values are A[99] = 198.000000000000000  B[99] = 99.000000000000000  C[99] = 297.000000000000000

(N=600)
Time in milliseconds= 0.053000000000000
Last Values are A[599] = 1198.000000000000000  B[599] = 599.000000000000000  C[599] = 1797.000000000000000

(N=2000)
Time in milliseconds= 0.040000000000000
Last Values are A[1999] = 3998.000000000000000  B[1999] = 1999.000000000000000  C[1999] = -259257304610248209780291474315334909952.000000000000000

Oh no! It's BROKEN! What can I do about this?

*/


// To compile: nvcc EbertHW1book.cu -O3 -o temp0 -lcudart
// To run: ./temp0

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

//Length of vectors to be added.
#define N 2000  //if N is greater than dimBlock.x program will break

float *A_CPU, *B_CPU, *C_CPU; //CPU pointers

float *A_GPU, *B_GPU, *C_GPU; //GPU pointers

dim3 dimBlock; //This variable will hold the Dimensions of your block

void AllocateMemory()
{					
	//Allocate Device (GPU) Memory, & allocates the value of the specific pointer/array
	hipMalloc(&A_GPU,N*sizeof(float));
	hipMalloc(&B_GPU,N*sizeof(float));
	hipMalloc(&C_GPU,N*sizeof(float));

	//Allocate Host (CPU) Memory
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));

}

//Loads values into vectors that we will add.
void Innitialize()
{
	int i;
	
	for(i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)2*i;	
		B_CPU[i] = (float)i;
	}
}

//Cleaning up memory after we are finished.
void CleanUp(float *A_CPU,float *B_CPU,float *C_CPU,float *A_GPU,float *B_GPU,float *C_GPU)  //free
{
	free(A_CPU); free(B_CPU); free(C_CPU);
	hipFree(A_GPU); hipFree(B_GPU); hipFree(C_GPU);
}

//This is the kernel. It is the function that will run on the GPU.
//It adds vectors A and B then stores result in vector C
__global__ void Addition(float *A, float *B, float *C, int n)
{

	int id = threadIdx.x;
	
	// This is the part that neds to be completed!
	C[id] = A[id] + B[id];  // Something like this?
}

int main()
{
	int i;
	timeval start, end; //The timeval structure is defined in sys/time.h
	hipError_t err;
	
	//Set the thread structure that you will be using on the GPU	
	// WHAT IS THIS FUNCTION?	
	//SetUpCudaDevices();

	//Partitioning off the memory that you will be using.
	AllocateMemory();

	//Loading up values to be added.
	Innitialize();
	
	//Starting the timer
	gettimeofday(&start, NULL);

	//Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	
	//Calling the Kernel (GPU) function.	
	Addition<<<1,N>>>(A_GPU, B_GPU, C_GPU, N);
	
	//Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);

	//Stopping the timer
	gettimeofday(&end, NULL);

	//Calculating the total time used in the addition and converting it to milliseconds.
	float time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	
	//Displaying the time 
	printf("Time in milliseconds= %.15f\n", (time/1000.0));	

	// Displaying vector info you will want to comment out the vector print line when your
	//vector becomes big. This is just to make sure everything is running correctly.	
	for(i = 0; i < N; i++)		
	{		
		//printf("A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", i, A_CPU[i], i, B_CPU[i], i, C_CPU[i]);
	}

	//Displaying the last value of the addition for a check when all vector display has been commented out.
	printf("Last Values are A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", N-1, A_CPU[N-1], N-1, B_CPU[N-1], N-1, C_CPU[N-1]);
	
	//You're done so cleanup your mess.
	CleanUp(A_CPU,B_CPU,C_CPU,A_GPU,B_GPU,C_GPU);	
	
	return(0);
}
