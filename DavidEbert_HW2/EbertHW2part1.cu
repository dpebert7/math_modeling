/*
David Ebert
Homework 2 - Part 1 - GPU Addition
*/

/*
For assignment 2 part 1, do the following:
	-Take the GPU vector addition program from HW1 and make it work with multiple blocks.
	-You will enter a number N which will be given to you to setup only just enough blocks to work with.
	-Add the vectors as normal.

The code below works for N values up to at least 10 million.

Printed results for N = 10 000:
Time in milliseconds= 0.090000000000000
Last Values are A[9999] = 9999.000000000000000  B[9999] = 9999.000000000000000  C[9999] = 19998.000000000000000\
*/


// To compile and run: nvcc EbertHW2part1.cu -O3 -o temp -lcudart -run
// To run: ./temp

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

//Length of vectors to be added.
#define N 10000

float *A_CPU, *B_CPU, *C_CPU; //CPU pointers

float *A_GPU, *B_GPU, *C_GPU; //GPU pointers

dim3 dimBlock; //This variable will hold the Dimensions of your block

void AllocateMemory()
{					
	//Allocate Device (GPU) Memory, & allocates the value of the specific pointer/array
	hipMalloc(&A_GPU,N*sizeof(float));
	hipMalloc(&B_GPU,N*sizeof(float));
	hipMalloc(&C_GPU,N*sizeof(float));

	//Allocate Host (CPU) Memory
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));

}

//Loads values into vectors that we will add.
void Innitialize()
{
	int i;
	
	for(i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)i;	
		B_CPU[i] = (float)i;
	}
}

//Cleaning up memory after we are finished.
void CleanUp(float *A_CPU,float *B_CPU,float *C_CPU,float *A_GPU,float *B_GPU,float *C_GPU)  //free
{
	free(A_CPU); free(B_CPU); free(C_CPU);
	hipFree(A_GPU); hipFree(B_GPU); hipFree(C_GPU);
}

//This is the kernel. It is the function that will run on the GPU.
//It adds vectors A and B then stores result in vector C
__global__ void Addition(float *A, float *B, float *C, int n)
{
	int thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	if(thread_id< n){
		C[thread_id] = A[thread_id] + B[thread_id];
	}

}

int main()
{
	int i;
	timeval start, end;
	//cudaError_t err; // Not sure what this is. Turning it off.
	
	//Set the thread structure that you will be using on the GPU	
	//SetUpCudaDevices(); // Not sure what this function is. Turning it off.

	//Partitioning off the memory that you will be using.
	AllocateMemory();

	//Loading up values to be added.
	Innitialize();
	
	//Starting the timer
	gettimeofday(&start, NULL);

	//Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	
	//Calling the Kernel (GPU) function.	
	Addition<<<(N+1023)/1024,1024>>>(A_GPU, B_GPU, C_GPU, N);
	
	//Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);

	//Stopping the timer
	gettimeofday(&end, NULL);

	//Calculating the total time used in the addition and converting it to milliseconds.
	float time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	
	//Displaying the time 
	printf("Time in milliseconds= %.15f\n", (time/1000.0));	

	// Displaying vector info you will want to comment out the vector print line when your
	//vector becomes big. This is just to make sure everything is running correctly.	
	for(i = 0; i < N; i++)		
	{		
		//printf("A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", i, A_CPU[i], i, B_CPU[i], i, C_CPU[i]);
	}

	//Displaying the last value of the addition for a check when all vector display has been commented out.
	printf("Last Values are A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", N-1, A_CPU[N-1], N-1, B_CPU[N-1], N-1, C_CPU[N-1]);
	
	//You're done so cleanup your mess.
	CleanUp(A_CPU,B_CPU,C_CPU,A_GPU,B_GPU,C_GPU);	
	
	return(0);
}
